#include "hip/hip_runtime.h"
#include <stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include <time.h>
#include <windows.h>
const int MAX_ROW_THREADS = 32; // 32x32 ---> 1024 thread su GPU
#define MAX 30


//eseguita da device e richiamabile da host e definisce funzione kernel
__global__ void cudamatmat(float * A,float * B,float * C, int N, int M, int P){
//sono matrici quadrate uso solo N

 int idy = blockIdx.y*blockDim.y+threadIdx.y;
 int idx = blockIdx.x*blockDim.x+threadIdx.x;

    float tmpSum = 0;

    if (idy < N && idx < N) {
        // each thread computes one element of the block sub-matrix
        for (int k = 0; k < N; k++) {
            tmpSum += A[idy * N + k] * B[k * N + idx];
        }
    }
    C[idy * N + idx] += tmpSum;
}




int main(){
	
	float *A, *B, *C;
	float *A_gpu= NULL, *B_gpu=NULL, *C_gpu=NULL;
	float total_t;
	int size = sizeof(float);
	float dim = 8000;
	int i;
	int numBlocks = dim / MAX_ROW_THREADS;
	clock_t start,stop;
	float gflops;

	//Imposta il dispositivo da utilizzare per le esecuzioni GPU.
	hipSetDevice(0);

	//allocazione matrici
	A=(float*)malloc(size*dim*dim); 
	B=(float*)malloc(size*dim*dim);
	C=(float*)malloc(size*dim*dim);

	//popolamento matrici
	for(i=0;i<dim*dim;i++){
	 	A[i] = rand()*MAX;
	 	B[i] = rand()*MAX;
	 	C[i] = rand()*MAX;
	}

	//copia delle matrici sul device
    hipMemcpy(A_gpu,A,size*dim*dim,hipMemcpyHostToDevice);
	hipMemcpy(B_gpu,B,size*dim*dim,hipMemcpyHostToDevice);
	hipMemcpy(C_gpu,C,size*dim*dim,hipMemcpyHostToDevice);

	//Una configurazione definisce la griglia e il numero di threads per ogni blocco
	dim3 DimGrid(numBlocks, numBlocks);    //(dimensione / 32 )^2
	dim3 DimBlock(MAX_ROW_THREADS, MAX_ROW_THREADS, 1); //numero di threads per block (1024)

	printf("\nStart Time: \n\n");
	start=clock();

	//estensione sintassi c
	cudamatmat <<< DimGrid, DimBlock >>> (A_gpu, B_gpu, C_gpu, dim, dim, dim);
	
	//mettere la CPU in attesa della terminazione 
	//di tutte le operazioni in esecuzione sul device
	hipDeviceSynchronize();

	stop=clock();
	printf("\nStop Time: \n\n");
	total_t =((float)(stop-start))/CLOCKS_PER_SEC;
	gflops = 2.0 * dim * dim * dim /(total_t)/ 1.e9;
	hipMemcpy(C,C_gpu,size*dim*dim,hipMemcpyDeviceToHost);


	printf("\nRisultati:\n");
	printf("\nGflops: %f\n\n",gflops);
	printf("\nTime: %f\n\n",total_t);

	//deallocazione memoria device
	hipFree(A_gpu);
	hipFree(B_gpu);
	hipFree(C_gpu);
	system("PAUSE");
	
}
